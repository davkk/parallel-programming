#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define STEPS 10000
#define BLOCK_SIZE 16
#define IDX(x, y, L) ((y) * L + (x))

void random_init(int *grid, int size) {
  for (int i = 0; i < size * size; ++i) {
    grid[i] = (float)rand() / RAND_MAX < 0.5 ? 0 : 1;
  }
}

__global__ void evolveKernel(int *curr, int *next, int size) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= size || y >= size) {
    return;
  }

  int count = 0;
  for (int i = -1; i <= 1; i++) {
    for (int j = -1; j <= 1; j++) {
      if (i == 0 && j == 0) {
        continue;
      }
      int nb_idx = IDX(x + j, y + i, size);
      if (nb_idx > 0 && nb_idx < size * size) {
        count += curr[nb_idx];
      }
    }
  }

  int cell = curr[IDX(x, y, size)];
  if (cell == 1) {
    next[IDX(x, y, size)] = (count == 2 || count == 3) ? 1 : 0;
  } else {
    next[IDX(x, y, size)] = (count == 3) ? 1 : 0;
  }
}

int main(int argc, char **argv) {
  srand(2001);

  int size = atoi(argv[1]);

  int *h_grid = (int *)malloc(size * size * sizeof(int));
  int *h_temp = (int *)malloc(size * size * sizeof(int));

  random_init(h_grid, size);

  int *d_grid, *d_temp;
  hipMalloc((void **)&d_grid, size * size * sizeof(int));
  hipMalloc((void **)&d_temp, size * size * sizeof(int));

  hipMemcpy(d_grid, h_grid, size * size * sizeof(int), hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x,
               (size + dimBlock.y - 1) / dimBlock.y);

  int steps = STEPS;
  while (steps--) {
    evolveKernel<<<dimGrid, dimBlock>>>(d_grid, d_temp, size);

    int *swap = d_grid;
    d_grid = d_temp;
    d_temp = swap;
  }

  hipMemcpy(h_grid, d_grid, size * size * sizeof(int), hipMemcpyDeviceToHost);

  // for (int i = 0; i < size * size; ++i) {
  //   printf("%d ", h_grid[i]);
  // }
  // printf("\n");

  hipFree(d_grid);
  hipFree(d_temp);

  free(h_grid);
  free(h_temp);

  return 0;
}
